#include "../cuda.hpp"
namespace tobi {namespace cuda {
	bool IsSupportCuda()
	{
		hipError_t _status = hipSetDevice(0);
		if (_status != hipSuccess)
			return false;
		return true;
	}

}
}